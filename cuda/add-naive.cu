
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void add(int n, float *x, float *y) {
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void) {
  int N = 1 << 20;
  float *x, *y;

  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  add<<<1, 1>>>(N, x, y);

  hipDeviceSynchronize();

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(x);
  hipFree(y);
}
